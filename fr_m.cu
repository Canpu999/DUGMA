
#include <hip/hip_runtime.h>
#include <stdio.h>

void check(hipError_t error_id)
{
  if (error_id != hipSuccess) {
  printf("hipGetDeviceCount returned %d\n-> %s\n",(int)error_id,hipGetErrorString(error_id));
  printf("Result = FAIL\n");
  exit(EXIT_FAILURE);
  }
}

void fr_m(
        float *Ptr[6]
        )

{ 

  // free device memory
    // free temporal device global memory
  check(hipFree(Ptr[0]));
  check(hipFree(Ptr[1]));
  check(hipFree(Ptr[2]));
  check(hipFree(Ptr[3]));
  check(hipFree(Ptr[4]));
  check(hipFree(Ptr[5]));
  check(hipDeviceReset());
  printf("Succeed in freeing the device memory \n");
  return;
}
